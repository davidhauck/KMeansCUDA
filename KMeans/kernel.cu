#include "hip/hip_runtime.h"
//David Hauck
//December 6, 2014

#include <stdio.h>
#include <cassert>

#include <float.h>

#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>

// CUDA helper functions
#include <hip/hip_runtime_api.h>

#define N 1000000
#define BLOCK_SIZE 32
#define K 4
#define BLOCK_SIZE_CLUSTERS 256

void runKMeansCUDA(int argc, char **argv);
void runKMeansCPU();

//dont need to sqrt, just seeing if a value is larger than others
__device__ double distance2(double x1, double y1, double x2, double y2)
{
	double x = x2 - x1;
	double y = y2 - y1;
	return x*x + y*y;
}

//dont need to sqrt, just seeing if a value is larger than others
double distance2CPU(double x1, double y1, double x2, double y2)
{
	double x = x2 - x1;
	double y = y2 - y1;
	return x*x + y*y;
}

__global__ void calcDistances(double* x, double* y, double* global_xNodes, double* global_yNodes, int* chosenNodes, int* changedNodes)
{
	int id = threadIdx.x;
	int block_offset = blockIdx.x*blockDim.x;

	//shared memory is about 100x faster than global. Its worth it to copy to shared memory
	__shared__ double xs[BLOCK_SIZE];
	__shared__ double ys[BLOCK_SIZE];
	__shared__ double xNodes[K];
	__shared__ double yNodes[K];
	__shared__ int localChangedNodes[BLOCK_SIZE];

	//copying over from global to shared memory goes faster on just one thread per block
	if (threadIdx.x == 0)
	{
		//copy coordinates to shared memory
		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			ys[i] = y[block_offset + i];
			xs[i] = x[block_offset + i];
		}
		//copy cluster centers to shared memory
		for (int i = 0; i < K; i++)
		{
			xNodes[i] = global_xNodes[i];
			yNodes[i] = global_yNodes[i];
		}
	}

	//make sure all memory is copied locally
	__syncthreads();

	//check distances to all cluster nodes. Take the closest one
	double minDistance = distance2(xs[id], ys[id], xNodes[0], yNodes[0]);
	int chosenNode = 0;
	for (int i = 1; i < K; i++)
	{
		double dist = distance2(xs[id], ys[id], xNodes[i], yNodes[i]);
		if (dist < minDistance)
		{
			minDistance = dist;
			chosenNode = i;
		}
	}

	//count how many coordinates changed clusters
	localChangedNodes[id] = 0;
	if (chosenNodes[block_offset + id] != chosenNode)
		localChangedNodes[id] = 1;
	chosenNodes[block_offset + id] = chosenNode;
	__syncthreads();
	for (int i = BLOCK_SIZE / 2; i > 0; i /= 2)
	{
		if (id < i)
		{
			localChangedNodes[id] += localChangedNodes[id + i];
		}
		__syncthreads();
	}
	changedNodes[blockIdx.x] = localChangedNodes[0];
}

__global__ void calcClusterCenters(double* x, double* y, double* global_xNodes, double* global_yNodes, int* chosenNodes)
{
	//each block will do one cluster
	int id = threadIdx.x;
	int k = blockIdx.x;
	__shared__ double xTotal[BLOCK_SIZE_CLUSTERS];
	__shared__ double yTotal[BLOCK_SIZE_CLUSTERS];
	__shared__ int numElements[BLOCK_SIZE_CLUSTERS];
	xTotal[id] = 0.;
	yTotal[id] = 0.;
	numElements[id] = 0.;

	//add up all of the elements belonging to a certain node
	for (int i = id; i < N; i += BLOCK_SIZE_CLUSTERS)
	{
		int chosenNode = chosenNodes[i];
		if (chosenNode == k)
		{
			xTotal[id] += x[i];
			yTotal[id] += y[i];
			numElements[id]++;
		}
	}

	//add together all of the values the nodes in the block found
	__syncthreads();
	for (int i = BLOCK_SIZE_CLUSTERS / 2; i > 0; i /= 2)
	{
		if (id < i)
		{
			xTotal[id] += xTotal[id + i];
			yTotal[id] += yTotal[id + i];
			numElements[id] += numElements[id + i];
		}
		__syncthreads();
	}

	//have one thread update the value with the new cluster center
	if (id == 0)
	{
		double newXPos = -1, newYPos = -1;
		if (numElements[0] != 0)
		{
			newXPos = xTotal[0] / numElements[0];
			newYPos = yTotal[0] / numElements[0];
		}
		global_xNodes[k] = newXPos;
		global_yNodes[k] = newYPos;
	}
}

double my_max(double n1, double n2)
{
	if (n1 > n2)
		return n1;
	return n2;
}

int main(int argc, char **argv)
{
	runKMeansCPU();
	runKMeansCUDA(argc, argv);
	hipDeviceReset();
	char s[100];
	fgets(s, sizeof(s), stdin);
}

int calcDistancesCPU(double* xCoords, double* yCoords, double* xNodes, double* yNodes, int* chosenNodes)
{
	int changedNodes = 0;
	//for every node, find the closest cluster center
	for (int i = 0; i < N; i++)
	{
		//loop through every cluster center to find the closest
		double minDistance = distance2CPU(xCoords[i], yCoords[i], xNodes[0], yNodes[0]);
		int chosenNode = 0;
		for (int j = 1; j < K; j++)
		{
			double dist = distance2CPU(xCoords[i], yCoords[i], xNodes[j], yNodes[j]);
			if (dist < minDistance)
			{
				minDistance = dist;
				chosenNode = j;
			}
		}

		//check if the node changed clusters
		if (chosenNodes[i] != chosenNode)
			changedNodes++;
		chosenNodes[i] = chosenNode;
	}
	return changedNodes;
}

void calcClusterCentersCPU(double* xCoords, double* yCoords, double* xNodes, double* yNodes, int* chosenNodes)
{
	double* xTotals = (double*)malloc(K * sizeof(double));
	double* yTotals = (double*)malloc(K * sizeof(double));
	int* numElements = (int*)malloc(K * sizeof(int));

	for (int i = 0; i < K; i++)
	{
		xTotals[i] = 0;
		yTotals[i] = 0;
		numElements[i] = 0;
	}

	//add up all the x and y values of all the coordinates for each cluster
	for (int i = 0; i < N; i++)
	{
		//check which cluster the coordinate belongs to
		int chosenNode = chosenNodes[i];
		//add the values to the corresponding cluster
		xTotals[chosenNode] += xCoords[i];
		yTotals[chosenNode] += yCoords[i];
		numElements[chosenNode]++;
	}

	//recalculate the centers
	for (int i = 0; i < K; i++)
	{
		//get the total sums of all the x and y coordinates
		double xTotal = xTotals[i];
		double yTotal = yTotals[i];
		//get the total number of elements
		int numE = numElements[i];
		if (numE > 0)
		{
			//find the average x and y coordinate
			double newX = xTotal / numE;
			double newY = yTotal / numE;
			//update the cluster center with its new value
			xNodes[i] = newX;
			yNodes[i] = newY;
		}
		else
		{
			xNodes[i] = -1;
			yNodes[i] = -1;
		}
	}
	free(xTotals);
	free(yTotals);
}

void runKMeansCPU()
{
	float solveTotal = 0, total = 0;
	for (int j = 0; j < 3; j++)
	{
		printf("CPU iteration %d:\r\n", j);
		hipEvent_t t1, t2, t3;
		hipEventCreate(&t1);
		hipEventCreate(&t2);
		hipEventCreate(&t3);


		hipEventRecord(t1, 0);
		hipEventSynchronize(t1);

		//initialize the data with random values
		double* xCoords = (double*)malloc(N * sizeof(double));
		double* yCoords = (double*)malloc(N * sizeof(double));
		for (int i = 0; i < N; i++)
		{
			xCoords[i] = rand() % 100;
			yCoords[i] = rand() % 100;
		}

		double* yNodes = (double*)malloc(K * sizeof(double));
		double* xNodes = (double*)malloc(K * sizeof(double));
		for (int i = 0; i < K; i++)
		{
			xNodes[i] = rand() % 100;
			yNodes[i] = rand() % 100;
		}

		int* chosenNodes = (int*)malloc(N * sizeof(int));
		for (int i = 0; i < N; i++)
		{
			chosenNodes[i] = -1;
		}

		hipEventRecord(t2, 0);
		hipEventSynchronize(t2);

		//keep iterating until less that 1% of the coordinates change clusters
		int changedNodes;
		bool shouldContinue;
		do
		{
			changedNodes = calcDistancesCPU(xCoords, yCoords, xNodes, yNodes, chosenNodes);
			calcClusterCentersCPU(xCoords, yCoords, xNodes, yNodes, chosenNodes);
			//printf("%d\r\n", changedNodes);
			shouldContinue = false;

			//keep looping until all clusters have at least one node (Not doing this in cuda do to memory copying overhead. When timing the runs, this is taken out.)
			/*for (int i = 0; i < K; i++)
			{
				if (xNodes[i] == -1)
				{
					shouldContinue = true;
					xNodes[i] = rand() % (int)maxX;
					yNodes[i] = rand() % (int)maxY;
				}
			}*/
			int x = 0;
		} while (changedNodes > 0.01 * N || shouldContinue);

		//free(xCoords);
		//free(yCoords);
		free(chosenNodes);

		hipEventRecord(t3, 0);
		hipEventSynchronize(t3);

		float timeCreate, timeSolve, timeTotal;
		hipEventElapsedTime(&timeCreate, t1, t2);
		hipEventElapsedTime(&timeSolve, t2, t3);
		hipEventElapsedTime(&timeTotal, t1, t3);
		printf("Create Time:%3.1f\r\nSolve Time:%3.1f\r\nTotal Time:%3.1f\r\n", timeCreate, timeSolve, timeTotal);
		printf("Cluster Centers:\r\n");
		for (int i = 0; i < K; i++)
		{
			printf("(%3.1f,\t%3.1f)\r\n", xNodes[i], yNodes[i]);
		}
		solveTotal += timeSolve;
		total += timeTotal;
		free(xNodes);
		free(yNodes);

	}
	float totalTimeAvg = total / 10;
	float solveTimeAvg = solveTotal / 10;
	printf("CPU Solve Time Avg:%3.1f\r\nTotal Time Avg:%3.1f\r\n", solveTimeAvg, totalTimeAvg);
}

void runKMeansCUDA(int argc, char **argv)
{
	float solveTotal = 0, total = 0;
	for (int j = 0; j < 3; j++)
	{
		printf("GPU iteration %d:\r\n", j);
		hipEvent_t t1, t2, t3, t4, t5;
		hipEventCreate(&t1);
		hipEventCreate(&t2);
		hipEventCreate(&t3);
		hipEventCreate(&t4);
		hipEventCreate(&t5);

		hipEventRecord(t1, 0);
		hipEventSynchronize(t1);
		//calculate the number of blocks based on the size of N. the block size is fixed at 32
		int Nblocks = N / 32;
		if (N % BLOCK_SIZE != 0)
		{
			Nblocks++;
		}
		int Nthreads = BLOCK_SIZE;

		//initialize the data with random values
		double* xCoords = (double*)malloc(N * sizeof(double));
		double* yCoords = (double*)malloc(N * sizeof(double));
		for (int i = 0; i < N; i++)
		{
			xCoords[i] = rand() % 100;
			yCoords[i] = rand() % 100;
		}

		double* yNodes = (double*)malloc(K * sizeof(double));
		double* xNodes = (double*)malloc(K * sizeof(double));
		for (int i = 0; i < K; i++)
		{
			xNodes[i] = rand() % 100;
			yNodes[i] = rand() % 100;
		}

		int* chosenNodes = (int*)malloc(N * sizeof(int));
		chosenNodes[0] = 16;
		for (int i = 1; i < N; i++)
		{
			chosenNodes[i] = -1;
		}

		int* changedNodes = (int*)malloc(Nblocks * sizeof(int));
		for (int i = 0; i < Nblocks; i++)
		{
			changedNodes[i] = -1;
		}

		hipEventRecord(t2, 0);
		hipEventSynchronize(t2);

		int devID;

		hipError_t error;
		hipDeviceProp_t deviceProp;

		devID = findCudaDevice(argc, (const char **)argv);

		checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

		if (deviceProp.major < 2)
		{
			hipDeviceReset();
			exit(EXIT_SUCCESS);
		}

		//create the matching data on the gpu
		double* d_xNodes;
		checkCudaErrors(hipMalloc((void **)&d_xNodes, K * sizeof(double)));
		double* d_yNodes;
		checkCudaErrors(hipMalloc((void **)&d_yNodes, K * sizeof(double)));

		double* d_xCoords;
		checkCudaErrors(hipMalloc((void **)&d_xCoords, N * sizeof(double)));
		double* d_yCoords;
		checkCudaErrors(hipMalloc((void **)&d_yCoords, N * sizeof(double)));

		int* d_chosenNodes;
		checkCudaErrors(hipMalloc((void **)&d_chosenNodes, N * sizeof(int)));

		int* d_changedNodes;
		checkCudaErrors(hipMalloc((void **)&d_changedNodes, Nblocks * sizeof(int)));

		//send the data to the gpu
		checkCudaErrors(hipMemcpy(d_xNodes, xNodes, K * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_yNodes, yNodes, K * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_xCoords, xCoords, N * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_yCoords, yCoords, N * sizeof(double), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_chosenNodes, chosenNodes, N * sizeof(int), hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(d_changedNodes, changedNodes, Nblocks * sizeof(int), hipMemcpyHostToDevice));


		// Kernel configuration, where a one-dimensional
		// grid and one-dimensional blocks are configured.
		dim3 dimGrid(Nblocks);
		dim3 dimBlock(Nthreads);

		dim3 kDim3(K);
		dim3 bscDim3(BLOCK_SIZE_CLUSTERS);

		hipEventRecord(t3, 0);
		hipEventSynchronize(t3);

		//keep iterating until less than 1% of the data changes clusters
		int totalChanges;
		do
		{
			calcDistances <<<dimGrid, dimBlock>>>(d_xCoords, d_yCoords, d_xNodes, d_yNodes, d_chosenNodes, d_changedNodes);
			checkCudaErrors(hipMemcpy(changedNodes, d_changedNodes, Nblocks * sizeof(int), hipMemcpyDeviceToHost));
			totalChanges = 0;
			for (int i = 0; i < Nblocks; i++)
			{
				totalChanges += changedNodes[i];
			}
			calcClusterCenters <<<kDim3, bscDim3 >>>(d_xCoords, d_yCoords, d_xNodes, d_yNodes, d_chosenNodes);
			printf("%d\r\n", totalChanges);
		} while (totalChanges > 0.01 * N);

		hipEventRecord(t4, 0);
		hipEventSynchronize(t4);

		double* newyNodes = (double*)malloc(K * sizeof(double));
		double* newxNodes = (double*)malloc(K * sizeof(double));
		checkCudaErrors(hipMemcpy(newxNodes, d_xNodes, K * sizeof(double), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(newyNodes, d_yNodes, K * sizeof(double), hipMemcpyDeviceToHost));
		hipFree(d_changedNodes);
		hipFree(d_chosenNodes);
		hipFree(d_xCoords);
		hipFree(d_xNodes);
		hipFree(d_yCoords);
		hipFree(d_yNodes);
		//free(xCoords);
		//free(yCoords);
		free(xNodes);
		free(yNodes);
		free(chosenNodes);
		free(changedNodes);
		hipEventRecord(t5, 0);
		hipEventSynchronize(t5);

		float timeCreate, timeSetup, timeSolve, timeFree, timeTotal;
		hipEventElapsedTime(&timeCreate, t1, t2);
		hipEventElapsedTime(&timeSetup, t2, t3);
		hipEventElapsedTime(&timeSolve, t3, t4);
		hipEventElapsedTime(&timeFree, t4, t5);
		hipEventElapsedTime(&timeTotal, t1, t5);
		printf("Create Time:%3.1f\r\nCuda Setup Time:%3.1f\r\nSolve Time:%3.1f\r\nDownload Answer and Free Memory Time:%3.1f\r\nTotal Time:%3.1f\r\n", timeCreate, timeSetup, timeSolve, timeFree, timeTotal);
		printf("Cluster Centers:\r\n");

		for (int i = 0; i < K; i++)
		{
			printf("(%3.1f,\t%3.1f)\r\n", newxNodes[i], newyNodes[i]);
		}
		solveTotal += timeSolve;
		total += timeTotal;
	}
	//free(xCoords);
	//free(yCoords);
	float totalTimeAvg = total / 10;
	float solveTimeAvg = solveTotal / 10;
	printf("GPU Solve Time Avg:%3.1f\r\nTotal Time Avg:%3.1f\r\n", solveTimeAvg, totalTimeAvg);
}
